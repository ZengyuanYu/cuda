
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

void __global__ add(const double *x, const double *y, double *z);
void check(const double *z, const int N);

int main() {
    const int N = 10000000;
    const int M = sizeof(double) * N;
    double *host_x = (double*)malloc(M);
    double *host_y = (double*)malloc(M);
    double *host_z = (double*)malloc(M);

    for (int n = 0; n < N; ++n) {
        host_x[n] = a;
        host_y[n] = b;
    }

    double *device_x, *device_y, *device_z;
    // cudaMalloc(void** adress, size_t size);
    hipMalloc((void **)&device_x, M);
    hipMalloc((void **)&device_y, M);
    hipMalloc((void **)&device_z, M);

    hipMemcpy(device_x, host_x, M, hipMemcpyHostToDevice);
    hipMemcpy(device_y, host_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size  = N / block_size;
    add<<<grid_size, block_size>>>(device_x, device_y, device_z); 
    hipMemcpy(host_z, device_z, M, hipMemcpyDeviceToHost);
    check(host_z, N);

    free(host_x);
    free(host_y);
    free(host_z);
    
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_z);

    return 0;
}

void __global__ add(const double *x, const double *y, double *z) {
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    z[n] = x[n] + y[n];
}

void check(const double *z, const int N) {
    bool has_error = false;
    for (int n = 0; n < N; ++n) {
        if (fabs(z[n] - c) > EPSILON) {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}